#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 
#include <driver_functions.h>

#include "CycleTimer.h"
#include "graph.h"
#include "sequential.h"       
#include "jp.h"           

#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <vector>
#include <set>
#include <random>
#include <chrono>
#include <algorithm> 

static std::mt19937 rng(std::random_device{}());
using namespace std;

// generate an erdos-renyi random undirected graph using a compressed‐sparse‐row representation

void random_graph_generator(int N, double p, vector<int> &adjacency_list, vector<int>& vertex_offsets){
    vector<vector<int>> neighbors(N);
    uniform_real_distribution<> dist(0.0, 1.0);
    // mt19937 rng(random_device{}());

    // add each edge (i,j) with probability ~p
    for (int i=0; i< N; i++){
        for (int j= i+1; j <N;j++){
            if (dist(rng) < p){
                neighbors[i].push_back(j);
                neighbors[j].push_back(i);

            }
        }
    }

    // find the offset for each vertex to populate the vertex offsets vextors
    // e.g vertex_offsets[i] = index in adjacency_list vector where all of vertex i's neighbors are listed
    // If adjacency_list = [3,7,2,0,5,6], and vertex_offsets = [0,2,3,6]
    // We have vertex 0's neighbors as 3 and 7
    // vertex 1 neighbors: 2
    // vertex 2 neighbors: 0,5,6

    vertex_offsets.resize(N+1);
    int idx = 0;
    for (int i=0; i <N;i++){
        vertex_offsets[i] = idx;
        for (int nei: neighbors[i]){
            adjacency_list.push_back(nei);
            idx++; 
        }
    }
    // the last value of vertex_offsets is the len of adjacency_list
    vertex_offsets[N] = idx;
}

__global__ void initial_jones_plassmann(int N, int* colors_device, bool* uncolored){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        colors_device[i] = 0;
        //vertex i is uncolored
        uncolored[i] = true; 
    }
}

__global__ void color_independent_set(int N, int* adjacency_list, int* vertex_offsets, int* priorities, int* colors, bool* uncolored, int* complete) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N || !uncolored[i]){
        return;
    }
    int curr_priority = priorities[i];
    for (int edge = vertex_offsets[i]; edge < vertex_offsets[i+1];edge++){
        int nei = adjacency_list[edge];
        if (uncolored[nei] && priorities[nei] > curr_priority){
            return;
        }
    }
    int color =1;
    bool conflict;

    for (color = 1;; color++){
        conflict = false;
        for (int edge = vertex_offsets[i]; edge < vertex_offsets[i+1];edge++){
            if (colors[adjacency_list[edge]] == color){
                conflict = true;
                break;
            }
        }
        if (!conflict){
            break;
        }
    }
    colors[i] = color;
    uncolored[i] = false;
    *complete = 1; 

}
    
void cuda_jones_plassmann(int N, int* adjacency_list, int* vertex_offsets, int* priorities, int* colors, bool* uncolored){
    int blockWidth = 128;
    dim3 blockDim(blockWidth,1,1);
    int gridWidth = (N + blockWidth -1)/(blockWidth);
    dim3 gridDim(gridWidth,1,1); 
    

    int* changed;
    hipMalloc(&changed, sizeof(int));

    while(true) {
        int has_changed = 0;
        hipMemcpy(changed, &has_changed, sizeof(int), hipMemcpyHostToDevice);
        color_independent_set<<<gridDim,blockDim>>>(N,adjacency_list,vertex_offsets,priorities, colors, uncolored, changed);
        hipDeviceSynchronize();
        hipMemcpy(&has_changed, changed, sizeof(int), hipMemcpyDeviceToHost);
        if (!has_changed){
            break;
        }
    }
    hipFree(changed);
}

void sequential_jones_plassmann(Graph& graph){
    int N = graph.size();
    graph.assign_priorities();
    std::vector<int>& colors = graph.get_colors();
    std::vector<int>& priorities = graph.get_priorities();

    bool colored_vertex = true;
    while (colored_vertex) {
        colored_vertex = false;
        // get all vertices to color
        std::vector<bool> uncolored;
        uncolored.resize(N, false);
        for (int u = 0; u < N; u++) {
            if (colors[u] == -1) {
                bool flag = true; // flag for if the vertex is local max
                std::vector<int> neighbors = graph.get_neighbors(u);
                for (int i = 0; i < neighbors.size(); i++) {
                    int v = neighbors[i];
                    if (colors[v] == -1 && priorities[v] > priorities[u]) {
                        flag = false;
                        break;
                    }
                }
                if (flag) {
                    uncolored[u] = true;
                }
            }
        }

        for (int u = 0; u < graph.size(); u++) {
            if (uncolored[u]) {
                std::set<int> nbor_colors;
                std::vector<int> neighbors = graph.get_neighbors(u);
                for (int i = 0; i < neighbors.size(); i++) {
                    int v = neighbors[i];
                    if (colors[v] != -1) {
                        nbor_colors.insert(colors[v]);
                    }
                }

                int color = 0;
                
                while (nbor_colors.count(color) > 0) { 
                    color++;
                }
                colors[u] = color;
                colored_vertex = true;
            }
        }
    }   
}


int main(){
    srand(time(nullptr)); 
    int N = 1000;
    double p = 0.01;

    vector<int> adjacency_list, vertex_offsets;
    random_graph_generator(N,p, adjacency_list, vertex_offsets);
    printf("Number of nodes: %d \nNumber of edges: %zu\n", N, adjacency_list.size()/2);

    int *adj_list_device, *vertex_offsets_device, *priorities_device, *colors_device; 
    bool* uncolored; 
    hipMalloc(&adj_list_device, adjacency_list.size()  * sizeof(int));
    hipMalloc(&vertex_offsets_device, vertex_offsets.size() * sizeof(int));
    hipMalloc(&priorities_device, N  * sizeof(int));
    hipMalloc(&colors_device, N * sizeof(int));
    hipMalloc(&uncolored, N * sizeof(bool));

    hipMemcpy(adj_list_device, adjacency_list.data(), adjacency_list.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(vertex_offsets_device, vertex_offsets.data(), vertex_offsets.size()  * sizeof(int), hipMemcpyHostToDevice);

    Graph graph(N); 
    for (int i=0; i<N; i++){
        for (int edge = vertex_offsets[i]; edge<vertex_offsets[i+1]; edge++){
            graph.add_edge(i, adjacency_list[edge]);
        }
    }
    graph.assign_priorities(); 
    vector<int> priorities = graph.get_priorities(); 
    hipMemcpy(priorities_device,priorities.data(), N * sizeof(int), hipMemcpyHostToDevice);

    int blockWidth = 128;
    dim3 blockDim(blockWidth,1,1);
    int gridWidth = (N + blockWidth -1)/(blockWidth);
    dim3 gridDim(gridWidth,1,1); 

    initial_jones_plassmann<<<gridDim, blockDim>>>(N,colors_device,uncolored);
    hipDeviceSynchronize();
    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    cuda_jones_plassmann(N,adj_list_device, vertex_offsets_device,priorities_device,colors_device, uncolored);
    hipEventRecord(end);
    hipEventSynchronize(end);
    float GPU_elapsed_time = 0;
    hipEventElapsedTime(&GPU_elapsed_time,start,end);


    vector<int> GPU_colors(N); 
    hipMemcpy(GPU_colors.data(),colors_device, N*sizeof(int), hipMemcpyDeviceToHost); 
    auto &colors = graph.get_colors();
    colors = GPU_colors; 

    bool valid_coloring = (graph.check_coloring() == 1);
    int num_colors_used = count_colors(graph); 

    printf("Jones-Plassman on GPU: valid:%d \ncolors:%d \n time elapsed: %.5f seconds\n", valid_coloring,num_colors_used, GPU_elapsed_time); 

    graph.reset_colors(); 
    double initial_time= CycleTimer::currentSeconds();
    sequential_jones_plassmann(graph); 
    double final_time = CycleTimer::currentSeconds(); 
    double elapsed_time = (final_time-initial_time);

    printf("Jones-Plassman on CPU: valid:%d \ncolors:%d \n time elapsed: %.5f seconds\n", valid_coloring,num_colors_used, elapsed_time); 

    graph.reset_colors(); 
    initial_time = CycleTimer::currentSeconds();
    greedy_color(graph); 
    final_time = CycleTimer::currentSeconds();  
    elapsed_time = (final_time-initial_time);

    printf("Greedy on CPU: valid:%d \ncolors:%d \n time elapsed: %.5f seconds\n", valid_coloring,num_colors_used, elapsed_time); 

    hipFree(adj_list_device);
    hipFree(vertex_offsets_device); 
    hipFree(priorities_device);
    hipFree(colors_device);
    hipFree(uncolored);
    return 0; 
}













