#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 
#include <driver_functions.h>

#include "CycleTimer.h"
#include "graph.h"
#include "sequential.h"       
#include "jp.h"           
#include "jp_cuda.h"
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <set>
#include <random>
#include <unistd.h>


static std::mt19937 rng(std::random_device{}());
using namespace std;

// generate an erdos-renyi random undirected graph using a compressed‐sparse‐row representation (utilizes less memory than normal adjacency list)

void random_graph_generator(int N, double p, vector<int> &adjacency_list, vector<int>& vertex_offsets){
    vector<vector<int>> neighbors(N);
    uniform_real_distribution<> dist(0.0, 1.0);

    // add each edge (i,j) with probability ~p
    for (int i=0; i< N; i++){
        for (int j= i+1; j <N;j++){
            if (dist(rng) < p){
                neighbors[i].push_back(j);
                neighbors[j].push_back(i);

            }
        }
    }

    // find the offset for each vertex to populate the vertex offsets vextors
    // e.g vertex_offsets[i] = index in adjacency_list vector where all of vertex i's neighbors are listed
    // If adjacency_list = [3,7,2,0,5,6], and vertex_offsets = [0,2,3,6]
    // We have vertex 0's neighbors as 3 and 7
    // vertex 1 neighbors: 2
    // vertex 2 neighbors: 0,5,6

    vertex_offsets.resize(N+1);
    int idx = 0;
    for (int i=0; i <N;i++){
        vertex_offsets[i] = idx;
        for (int nei: neighbors[i]){
            adjacency_list.push_back(nei);
            idx++; 
        }
    }
    // the last value of vertex_offsets is the len of adjacency_list
    vertex_offsets[N] = idx;
}


// GPU Kernel function that initalizes the data structures Jones-plassman uses to color the graph 
__global__ void initial_jones_plassmann(int N, int* colors_device, bool* uncolored){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        colors_device[i] = 0;
        uncolored[i] = true; 
    }
}


// GPU kernel function that does an iteration of jones-plassman, coloring an maximal independent set of vertices
__global__ void color_independent_set(int N, int* adjacency_list, int* vertex_offsets, int* priorities, int* colors, bool* uncolored, int* complete) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N || !uncolored[i]){
        return;
    }
    int curr_priority = priorities[i];
    for (int edge = vertex_offsets[i]; edge < vertex_offsets[i+1];edge++){
        int nei = adjacency_list[edge];
        if (uncolored[nei] && priorities[nei] > curr_priority){
            return;
        }
    }
    int color =1;
    bool flag;

    for (color = 1;; color++){
        flag = false;
        for (int edge = vertex_offsets[i]; edge < vertex_offsets[i+1];edge++){
            if (colors[adjacency_list[edge]] == color){
                flag = true;
                break;
            }
        }
        if (!flag){
            break;
        }
    }
    colors[i] = color;
    uncolored[i] = false;
    *complete = 1; 

}

// host code that calls the kernel functions
void cuda_jones_plassmann(int N, int num_threads, int* adjacency_list, int* vertex_offsets, int* priorities, int* colors, bool* uncolored){
    int blockWidth = num_threads;
    dim3 blockDim(blockWidth,1,1);
    int gridWidth = (N + blockWidth -1)/(blockWidth);
    dim3 gridDim(gridWidth,1,1); 
    

    int* changed;
    hipMalloc(&changed, sizeof(int));
    //calls kernel function that colors MIS for each JP iteration until entire graph has been colored
    while(true) {
        int has_changed = 0;
        hipMemcpy(changed, &has_changed, sizeof(int), hipMemcpyHostToDevice);
        color_independent_set<<<gridDim,blockDim>>>(N,adjacency_list,vertex_offsets,priorities, colors, uncolored, changed);
        hipDeviceSynchronize();
        hipMemcpy(&has_changed, changed, sizeof(int), hipMemcpyDeviceToHost);
        if (!has_changed){
            break;
        }
    }
    hipFree(changed);
}


int main(int argc, char** argv) {
    srand(time(nullptr)); 
    int opt;
    int num_vertices;
    int num_threads;
    double edge_prob;

    while ((opt = getopt(argc, argv, "v:p:n:")) != -1) {
        switch (opt) {
        case 'v':
            num_vertices = atoi(optarg);
            break;
        case 'p':
            edge_prob = atof(optarg);
            break;
        case 'n':
            num_threads = atoi(optarg);
            break;
        // case 'a':
        //     algo = optarg;
        //     break;
        default:
            std::cerr << "Usage: " << argv[0] << " -v num_vertices [-p edge_prob] -n num_threads\n";
            exit(EXIT_FAILURE);
        }
    }

    // Check if required options are provided
    if (num_vertices <= 0 || edge_prob < 0 || num_threads <= 0) {
        std::cerr << "Usage: " << argv[0] << " -v num_vertices [-p edge_prob] -n num_threads -a algorithm\n";
        exit(EXIT_FAILURE);
    }
    int N = num_vertices;
    double p = edge_prob;
    double initial_time, final_time, elapsed_time;
    initial_time= CycleTimer::currentSeconds();



    vector<int> adjacency_list, vertex_offsets;
    
    random_graph_generator(N,p, adjacency_list, vertex_offsets);
    final_time = CycleTimer::currentSeconds(); 
    elapsed_time = (final_time-initial_time);
    printf("Initalization time: %.5f \n", elapsed_time);

    printf("Number of nodes: %d \nNumber of edges: %zu\n", N, adjacency_list.size()/2);
    printf("__________________________________________\n");
    int *adj_list_device, *vertex_offsets_device, *priorities_device, *colors_device; 
    bool* uncolored; 

    // Allocate GPU memory for graph data
    hipMalloc(&adj_list_device, adjacency_list.size()  * sizeof(int));
    hipMalloc(&vertex_offsets_device, vertex_offsets.size() * sizeof(int));
    hipMalloc(&priorities_device, N  * sizeof(int));
    hipMalloc(&colors_device, N * sizeof(int));
    hipMalloc(&uncolored, N * sizeof(bool));

    hipMemcpy(adj_list_device, adjacency_list.data(), adjacency_list.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(vertex_offsets_device, vertex_offsets.data(), vertex_offsets.size()  * sizeof(int), hipMemcpyHostToDevice);
    
    // Create Graph object for CPU algos
    Graph graph(N); 
    for (int i=0; i<N; i++){
        for (int edge = vertex_offsets[i]; edge<vertex_offsets[i+1]; edge++){
            graph.add_edge(i, adjacency_list[edge]);
        }
    }
    graph.assign_priorities(); 
    vector<int> priorities = graph.get_priorities(); 
    hipMemcpy(priorities_device,priorities.data(), N * sizeof(int), hipMemcpyHostToDevice);
    bool valid_coloring;
    int num_colors_used; 

    // kernel launch params
    int blockWidth = num_threads;
    dim3 blockDim(blockWidth,1,1);
    int gridWidth = (N + blockWidth -1)/(blockWidth);
    dim3 gridDim(gridWidth,1,1); 

    initial_time= CycleTimer::currentSeconds();
    initial_jones_plassmann<<<gridDim, blockDim>>>(N,colors_device,uncolored);
    hipDeviceSynchronize();
    cuda_jones_plassmann(N,num_threads, adj_list_device, vertex_offsets_device,priorities_device,colors_device, uncolored);
    final_time = CycleTimer::currentSeconds(); 

    double GPU_elapsed_time = (final_time-initial_time);;
    vector<int> GPU_colors(N); 
    hipMemcpy(GPU_colors.data(),colors_device, N*sizeof(int), hipMemcpyDeviceToHost); 
    auto &colors = graph.get_colors();
    colors = GPU_colors; 
    
    valid_coloring = (graph.check_coloring() == 1);
    num_colors_used = graph.count_colors(); 

    printf("Jones-Plassman on GPU:\nvalid coloring: %s\ncolors: %d \ntime elapsed: %.5f seconds\n", valid_coloring ? "true": "false",num_colors_used, GPU_elapsed_time); 
    printf("__________________________________________\n");

    graph.reset_colors(); 
    initial_time= CycleTimer::currentSeconds();
    jones_plassmann(graph); 
    final_time = CycleTimer::currentSeconds(); 
    elapsed_time = (final_time-initial_time);
    valid_coloring = (graph.check_coloring() == 1);
    num_colors_used = graph.count_colors(); 

    printf("Jones-Plassman on CPU:\nvalid coloring: %s\ncolors: %d \ntime elapsed: %.5f seconds\n", valid_coloring ? "true": "false",num_colors_used, elapsed_time); 
    printf("speedup(JP): %.5f\n", elapsed_time/GPU_elapsed_time);
    printf("__________________________________________\n");


    graph.reset_colors(); 
    initial_time = CycleTimer::currentSeconds();
    greedy_color(graph); 
    final_time = CycleTimer::currentSeconds();  
    elapsed_time = (final_time-initial_time);
    valid_coloring = (graph.check_coloring() == 1);
    num_colors_used = graph.count_colors(); 
    printf("Greedy on CPU:\nvalid coloring: %s\ncolors: %d \ntime elapsed: %.5f seconds\n", valid_coloring ? "true": "false",num_colors_used, elapsed_time); 
    printf("speedup(Greedy): %.5f\n", elapsed_time/GPU_elapsed_time);

    hipFree(adj_list_device);
    hipFree(vertex_offsets_device); 
    hipFree(priorities_device);
    hipFree(colors_device);
    hipFree(uncolored);
    return 0; 
}
